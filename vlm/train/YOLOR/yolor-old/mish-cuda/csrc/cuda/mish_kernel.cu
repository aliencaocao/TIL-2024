#include <torch/types.h>
#include <hip/hip_runtime.h>
#include "CUDAApplyUtils.cuh"

// TORCH_CHECK replaces AT_CHECK in PyTorch 1,2, support 1.1 as well.
#ifndef TORCH_CHECK
#define TORCH_CHECK AT_CHECK
#endif

#ifndef __CUDACC_EXTENDED_LAMBDA__
#error "please compile with --expt-extended-lambda"
#endif

namespace kernel {
#include "mish_cuda.h"

using at::cuda::CUDA_tensor_apply2;
using at::cuda::CUDA_tensor_apply3;
using at::cuda::TensorArgType;

template <typename scalar_t>
void
mish_forward(
  torch::Tensor &output,
  const torch::Tensor &input
) {
  CUDA_tensor_apply2<scalar_t,scalar_t>(
    output, input,
    [=] __host__ __device__ (scalar_t &out, const scalar_t &inp) {
      mish_fwd_func(out, inp);
    },
    TensorArgType::ReadWrite, TensorArgType::ReadOnly
  );
}

template <typename scalar_t>
void
mish_backward(
  torch::Tensor &grad_inp,
  const torch::Tensor &input,
  const torch::Tensor &grad_out
) {
  CUDA_tensor_apply3<scalar_t,scalar_t,scalar_t>(
    grad_inp, input, grad_out,
    [=] __host__ __device__ (scalar_t &grad_inp, const scalar_t &inp, const scalar_t &grad_out) {
      mish_bwd_func(grad_inp, inp, grad_out);
    },
    TensorArgType::ReadWrite, TensorArgType::ReadOnly, TensorArgType::ReadOnly
  );
}

} // namespace kernel

void
mish_forward_cuda(
    torch::Tensor &output, const torch::Tensor &input
) {
  auto in_arg  = torch::TensorArg(input,  "input",  0),
       out_arg = torch::TensorArg(output, "output", 1);
  torch::checkAllDefined("mish_forward_cuda", {in_arg, out_arg});
  torch::checkAllSameGPU("mish_forward_cuda", {in_arg, out_arg});
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "mish_forward_cuda", [&] {
      kernel::mish_forward<scalar_t>(output, input);
  });
}

void
mish_backward_cuda(
  torch::Tensor &grad_inp, const torch::Tensor &input, const torch::Tensor &grad_out
) {
  auto gi_arg = torch::TensorArg(grad_inp, "grad_inp", 0),
       in_arg = torch::TensorArg(input,    "input",    1),
       go_arg = torch::TensorArg(grad_out, "grad_out", 2);
  torch::checkAllDefined("mish_backward_cuda", {gi_arg, in_arg, go_arg});
  torch::checkAllSameGPU("mish_backward_cuda", {gi_arg, in_arg, go_arg});
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_inp.scalar_type(), "mish_backward_cuda", [&] {
      kernel::mish_backward<scalar_t>(grad_inp, input, grad_out);
  });
}
